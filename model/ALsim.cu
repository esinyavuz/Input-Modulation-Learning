/*--------------------------------------------------------------------------
   Author: Thomas Nowotny
   Contributed by: Esin Yavuz
  
   Institute: Center for Computational Neuroscience and Robotics
              University of Sussex
              Falmer, Brighton BN1 9QJ, UK 
  
   email to:  T.Nowotny@sussex.ac.uk
  
   initial version: 2011-06-14
  
   This code is based on a previous model of the honeybee antennal lobe,
   used in the article by Nowotny T, Stierle JS, Galizia CG, Szyszka P. 2013
   "Data-driven honeybee antennal lobe model suggests how stimulus-onset
   asynchrony can aid odour segregation." Brain Res.
--------------------------------------------------------------------------*/

#include "AL.h"

#include "gauss.h"
#include "randomGen.h"
#include "randomGen.cc"
#include "standard_deviation.cc"
#include "hr_time.cpp"
randomGen R;
randomGauss RG;

#include "AL.cc"

CStopWatch timer;
unsigned int sumORN, sumPN, sumhLN, sumLHI =0;

int main(int argc, char *argv[])
{
  if (argc != 4) {
    cerr << "usage: ALsim <directory> <infile basename> <CPU= 0, GPU=1>" << endl;
    exit(1);
  }
  
  cerr << "% call was: ";
  for (int i= 0; i < argc; i++) {
    cerr << argv[i] << " ";
  }
  cerr << endl;

  unsigned int which= atoi(argv[3]);

  double tlast= 0.0, tlastwrite= 0.0; 

  stringstream sname;
  char thename[80];

  cerr << "% proto file: ";
  sname.clear();
  sname << argv[1] << "/" << argv[2];
  sname << ".proto" << ends;
  sname >> thename;
  ifstream pris(thename);

  cerr << "% out file: ";
  sname.clear();
  sname << argv[1] << "/" << argv[2];
  sname << ".out.cmp" << ends;
  sname >> thename;
  ofstream os(thename);
  os.precision(10);

  cerr << "% out file st: ";
  sname.clear();
  sname << argv[1] << "/" << argv[2];
  sname << ".out.st" << ends;
  sname >> thename;
	cerr << thename << " is thename" << endl;
  ofstream stos(thename);
  stos.precision(5);

  AL al(which);
	double tme;
  int cntrr= 0;

  cerr << "% odorPath: " << odorPath << endl;
  cerr << "% odorExtension: " << odorExtension << endl;
  cerr << "% LNPNsynFile: " <<  LNPNsynFile << endl;
  R.seedrand((unsigned long) seed, (unsigned long) seed+1, (unsigned long) seed+2);
  RG.seedrand((unsigned long) seed, (unsigned long) seed+1, (unsigned long) seed+2);  
  al.readOdors((int) _nOdor,odorPath, odorExtension);
  al.connect_ORN_PN1();
  al.connect_ORN_PN();
  al.connect_ORN_hLN();
  al.connect_PN_hLN();
  al.connect_PN_LHI();

  al.connect_hLN_PN(LNPNsynFile);
  al.connect_hLN_hLN();
  al.allocate_direct_input();
  al.read_protocol(pris);
  al.randomize_V();
  al.enable();
  if (write_raw || write_all) {
      if (write_raw) al.output_state(os);
      else {
	      al.output_matlab_helper_full(toString(argv[1])+"/"+toString(argv[2]));
	      al.output_full_state(os);
      }
  } 

  if ((int) readState) {
    sname.clear();
    sname << argv[1];
    sname << ".stateIn.bin" << ends;
    sname >> thename;
    ifstream sis(thename, ios::binary);
    assert(sis.good());
    al.read_state(sis);
  }

  al.output_state_st(stos);

  timer.startTimer();
  while (al.continues()) {
    if (write_raw || write_all) {
        if (t-tlastwrite > write_interval) {
            if (write_raw) al.output_state(os);	
	    else al.output_full_state(os);
	    tlastwrite= t;
        }
    }
    al.run();
    if (which == GPU) {
  copySpikeNFromDevice();
	copySpikesFromDevice();
    }
	sumORN+=spikeCount_ORN;
	sumPN+=spikeCount_PN;
	sumhLN+=spikeCount_hLN;
	sumLHI+=spikeCount_LHI;


    al.output_state_st(stos);
  }

  if ((int) writeState) {
      sname.clear();
      sname << argv[1];
      sname << ".stateOut.bin" << ends;
      sname >> thename;
      ofstream sis(thename, ios::binary);
      assert(sis.good());
      al.write_state(sis);
  }
  timer.stopTimer();
  stos.close();
  tme= timer.getElapsedTime();
  hipDeviceReset();
  cerr << "elapsed time: " << tme << ", " << sumORN << " ORN "<< sumPN << " PN " << sumhLN << " LN " << sumLHI << " LHI spikes." << endl;

  return 0;
}
  
